#include "hip/hip_runtime.h"
#include "StiffnessMatrixGPU.h"

__host__ __device__ void constantCreatorGPU(unsigned int numberElement, float* c, float* x, float* y, unsigned int* mesh)
{
  printf("i am here i is: %d\n", numberElement);
  unsigned int i = numberElement*6;
  c[i++] = (x[mesh[numberElement*4+0]] - x[mesh[numberElement*4+1]] + x[mesh[numberElement*4+2]] - x[mesh[numberElement*4+3]])/4; 
  c[i++] = (x[mesh[numberElement*4+0]] - x[mesh[numberElement*4+1]] - x[mesh[numberElement*4+2]] + x[mesh[numberElement*4+3]])/4;
  c[i++] = (x[mesh[numberElement*4+0]] - x[mesh[numberElement*4+3]] - x[mesh[numberElement*4+2]] + x[mesh[numberElement*4+1]])/4;
  c[i++] = (y[mesh[numberElement*4+0]] - y[mesh[numberElement*4+1]] + y[mesh[numberElement*4+2]] - y[mesh[numberElement*4+3]])/4;
  c[i++] = (y[mesh[numberElement*4+0]] - y[mesh[numberElement*4+1]] - y[mesh[numberElement*4+2]] + y[mesh[numberElement*4+3]])/4;
  c[i++] = (y[mesh[numberElement*4+0]] - y[mesh[numberElement*4+3]] - y[mesh[numberElement*4+2]] + y[mesh[numberElement*4+1]])/4;
  // defined the constants c1x to c3y
};


__global__ void constantCreator2(int n, float* c, float* x, float* y, unsigned int* mesh, StiffnessMatrixGPU *s)
{
  //printf("in the function\n blockDim.x = %d, gridDim.x = %d, blockIdx.x = %d\n", blockDim.x,gridDim.x, blockIdx.x);
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    {
      //printf("i is %d stride is %d threadID = %d\n",i,stride,threadIdx.x);
      //s->constantCreator(i, c, x, y, mesh);
    }
};

__host__ __device__
StiffnessMatrixGPU::StiffnessMatrixGPU(Material& mat, Geometry &geo, unsigned int n)
  : StiffnessMatrixFirstOrder(mat, geo, n)
{
#if __CUDA_ARCH__
  printf("Stiffness Matrix GPU Created by GPU");
#elif !defined(__CUDA_ARCH__)
  int device = -1;
  hipGetDevice(&device);
  // define array in GPU
  hipMallocManaged(&integrationNode_d, numberOfIntegrationPoint*sizeof(float));
  hipMallocManaged(&integrationPos_d, numberOfIntegrationPoint*dimention*numberOfIntegrationPoint*sizeof(unsigned int));
  hipMallocManaged(&integrationWeight_d, numberOfIntegrationPoint*sizeof(float));
  hipMemcpy(integrationNode_d, integrationNode, numberOfIntegrationPoint*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(integrationPos_d, integrationPos, numberOfIntegrationPoint*dimention*numberOfIntegrationPoint*sizeof(unsigned int), hipMemcpyHostToDevice);
  hipMemcpy(integrationWeight_d,integrationWeight, numberOfIntegrationPoint*sizeof(float), hipMemcpyHostToDevice);
  // copying arrays in geometry class to the gpu
  hipMallocManaged(&x_d, geometry->numberOfNodes*sizeof(float));
  hipMallocManaged(&y_d, geometry->numberOfNodes*sizeof(float));
  hipMallocManaged(&mesh_d, geometry->numberOfElementsG*4*sizeof(unsigned int));
  hipMemcpy(x_d, geometry->x, geometry->numberOfNodes*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(y_d, geometry->y, geometry->numberOfNodes*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(mesh_d, geometry->mesh, geometry->numberOfElementsG*4*sizeof(unsigned int), hipMemcpyHostToDevice);
  // constant stiffMat and DOFs
  hipMallocManaged(&c_d, numberOfElements*6*sizeof(float));
  hipMallocManaged(&stiffMat_d, stiffMatSize*sizeof(float));
  // copy from the material matarix
  hipMallocManaged(&D_d, 6*sizeof(float));
  hipMemcpy(D_d, material->materialMatrix, 6*sizeof(float), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  Log::Logger().Info("StiffnessMatrixGPU created by CPU");
#endif
};

__host__ __device__ StiffnessMatrixGPU::~StiffnessMatrixGPU()
{
#if __CUDA_ARCH__
  printf("Stiffness Matrix GPU deleted by GPU");
#elif !defined(__CUDA_ARCH__)
  Log::Logger().Info("StiffnessMatrixGPU deleted by CPU");
  hipFree(integrationPos_d);
  hipFree(integrationNode_d);
  hipFree(integrationWeight_d);
  hipFree(c_d);
  hipFree(stiffMat_d);
  hipFree(x_d);
  hipFree(y_d);
  hipFree(mesh_d);
  hipFree(D_d);
#endif
}

__host__ float* StiffnessMatrixGPU::GetStiffnessMatrix()
{
  blockSize = 32;
  //numberOfElements=33;
  int numBlocks = (numberOfElements + blockSize-1)/blockSize;
  constantCreator2<<<numBlocks, blockSize>>>(numberOfElements, c_d, x_d, y_d, mesh_d, this);
  hipDeviceSynchronize();
  return c_d;
}
