#include "hip/hip_runtime.h"
#include "Sparse.h"

Sparse::Sparse() {};

Sparse::Sparse(unsigned int x_size, unsigned int rowSize, unsigned int columnSize)
  : valueSize(x_size), numberOfRows(rowSize), numberOfColumns(columnSize)
{
  Log::Logger().Info("Sparse created");
  hipMallocManaged(&i, valueSize*sizeof(unsigned int));
  hipMallocManaged(&j, valueSize*sizeof(unsigned int));  
  hipMallocManaged(&value, valueSize*sizeof(float));
  hipMallocManaged(&nnz_inRow, numberOfRows*sizeof(unsigned int));
  hipMemset(j,0,valueSize*sizeof(unsigned int));
  hipMemset(i,0,valueSize*sizeof(unsigned int));
  hipMemset(value, 0 , valueSize*sizeof(float));
}

Sparse::Sparse(unsigned int x_size, unsigned int sizeOfMatrix)
  : Sparse(x_size, sizeOfMatrix, sizeOfMatrix)
{
  symmetry = true;
};

Sparse::~Sparse() {
  Log::Logger().Info("Sparse deleted");
  hipFree(i);
  hipFree(j);
  hipFree(value);
  hipFree(nnz_inRow);
}

// This function return a CSC matrix
void Sparse::Assemble()
// coo -> a triplet sparse format
{
  unsigned int * a_temp_i;  unsigned int * a_temp_j; float * a_temp_value;
  // build the indices vector
  unsigned int* indices = new unsigned int[valueSize];
  for ( unsigned int c = 0; c < valueSize; c++) { indices[c] = c;};
  // timer
  Timer timer("Time spend in assembley: ");
  // sorting i
  std::sort(indices,indices+valueSize, sort_indices(i));
  // sorting j
  unsigned int a = 0, b =0;
  for (unsigned int c = 0; c < numberOfRows; c++) {
    a = a + nnz_inRow[c]; b = a + nnz_inRow[c+1];
    std::sort(indices + a, indices+b, sort_indices_j(j,value));
  }
  

  // copy to new array
  hipMallocManaged(&a_temp_i, valueSize*sizeof(unsigned int));
  hipMallocManaged(&a_temp_j, valueSize*sizeof(unsigned int)); 
  hipMallocManaged(&a_temp_value,valueSize*sizeof(float));
  unsigned int vs = 0; // new value size
  for (unsigned int c = 0; c < valueSize; c++) {
    //if (abs(value[indices[c]]) > 0.001) {
    a_temp_i[vs] = i[indices[c]];
    a_temp_j[vs] = j[indices[c]];
    a_temp_value[vs] = value[indices[c]];
    vs++;
    //}
  }
  //valueSize = vs;
  hipFree(i); hipFree(j); hipFree(value);
  i = a_temp_i;
  j = a_temp_j;
  value = a_temp_value;
  
  //
  delete[] indices;
}
  /*  
  Log::Logger().Info(rowPtr,numberOfRows+1); 
  unsigned int counter = 0;
  for (unsigned int r = 1; r < numberOfRows; r++) {
    a = 0;
    for (unsigned int c = rowPtr[r]; c < rowPtr[r+1]; c++) {
      if (abs(value[indices[counter]]) > 0.001) {
	a++;
      }
      counter++;
    }
    rowPtr[r+1] = a;
  }
  // just change the variable for test
  unsigned int tempValueSize = valueSize; 
  //valueSize = rowPtr[numberOfRows];
  hipDeviceSynchronize;
  */

float Sparse::Compare(Sparse& A, Sparse& B)
// return the max Error between the two matrices
{
  float MaxError = 0.00;
  for (unsigned int counter = 0; counter < A.valueSize; counter++)
    MaxError = fmax(MaxError, fabs(A.value[counter] - B.value[counter]));
  return MaxError;
}

void Sparse::solver(Sparse& matrix, Sparse& vector) {
  Timer timer("Time spend in GPU for inverting: ");
  hipError_t cudaStatus;
  hipsolverStatus_t  cusolverStatus;
  hipsolverHandle_t  handle;
  float   *Work;                                         //   workspace
  int   *info , Lwork;                      //   info , workspace  size
  cudaStatus = hipGetDevice (0);
  cusolverStatus = hipsolverDnCreate (& handle ); //  create  handle
  hipblasFillMode_t  uplo = HIPBLAS_FILL_MODE_LOWER;
  hipMallocManaged (&info ,sizeof(int ));// unified  mem. for  info
  //  compute  workspace  size  and  prepare  workspace
  cusolverStatus = hipsolverDnSpotrf_bufferSize(handle , uplo ,matrix.numberOfRows,matrix.value,matrix.numberOfRows,& Lwork );
  hipMallocManaged (&Work ,Lwork*sizeof(float )); //mem.for  Work
  //  Cholesky  decomposition   d_A=L*L^T, lower  triangle  of d_A is
  //  replaced  by the  factor L
  cusolverStatus = hipsolverDnSpotrf(handle,uplo,matrix.numberOfRows,matrix.value,matrix.numberOfRows,Work,Lwork,info);
  cudaStatus = hipDeviceSynchronize ();
  //  solve A*X=B,   where A is  factorized  by  potrf  function
  // B is  overwritten  by the  solution
  cusolverStatus = hipsolverDnSpotrs(handle,uplo,matrix.numberOfRows,1,matrix.value,matrix.numberOfRows,vector.value,matrix.numberOfRows,info);
  cudaStatus = hipDeviceSynchronize ();
  cudaStatus = hipFree(info);
  cudaStatus = hipFree(Work);
  cusolverStatus = hipsolverDnDestroy(handle);
  //cudaStatus = hipDeviceReset();
}

// Setters
void Sparse::set_i(unsigned int* index_i) { i = index_i;}
void Sparse::set_j(unsigned int* index_j) { j = index_j;}
void Sparse::set_x(float* x) { value = x;}
void Sparse::set_numberOfRows(unsigned int x) { numberOfRows = x;}
void Sparse::set_valueSize(unsigned int x) { valueSize = x;}

void Sparse::set_numberOfColumns(unsigned int x) {
  numberOfColumns = x;
  if (x == 1 ) { // if we are dealing with vecotrs 
    j = new unsigned int[valueSize];
    for (int ci = 0; ci < valueSize; ci++) {
      j[ci] = 1;
    }
  }
}
  
// Geters
unsigned int Sparse::get_valueSize() const {return valueSize;};
unsigned int Sparse::get_numberOfRows() const { return numberOfRows;}
unsigned int Sparse::get_numberOfColumns() const {return numberOfColumns;}
float * Sparse::get_value() const {return value;}
unsigned int * Sparse::get_i() const {return i;}
unsigned int * Sparse::get_j() const {return j;}

// -- override the cout << oprator 
std::ostream& operator<< (std::ostream &out, Sparse const& sp) {
  const float* x = sp.get_value();
  const unsigned int* i = sp.get_i();
  const unsigned int* j = sp.get_j();
  for (int c = 0; c < sp.get_valueSize(); c++) {
    out << '\t' << i[c] << '\t' << j[c] << ':' << '\t' << x[c] << '\n';
  }
  return out;
}

// printers
void Sparse::Print() {
  std::cout << "\033[1;34m[SparseMatrix]: \033[0m" << numberOfRows  << " x " << numberOfColumns << " ";
  if (symmetry)
    std::cout << "\033[32msymmetry\033[0m " ;
  unsigned int size = valueSize <72 ? valueSize : 72; 
  std::cout << "print size: " << size << ", nnz = " << valueSize << std::endl;
  for (unsigned int counter = 0; counter < size; counter++)
    std::cout << i[counter] << "\t"<< j[counter] <<"\t: " << value[counter] <<std::endl;
};

unsigned int Sparse::Printer(Sparse& s)
{
  s.Print();
  return 0;
};

// ---------------------- sort_indices struct -------------------------
sort_indices::sort_indices(unsigned int* var)
  : dofSorted(var) {};

bool sort_indices::operator()(unsigned int i, unsigned int j) const { return dofSorted[i] < dofSorted[j];};

// ---------------------- sort_indices_j struct ------------------------- 

sort_indices_j::sort_indices_j(unsigned int* var, float* value)
  : dofSorted(var), x(value) {};

bool sort_indices_j::operator()(unsigned int i, unsigned int j) {
  if (dofSorted[i] == dofSorted[j]) {
    x[i] = x[i] + x[j]; x[j] = 0;
    return true;
  } else {return dofSorted[i] < dofSorted[j];}
};
















