#include "hip/hip_runtime.h"
#include "StiffnessMatrixGPU.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
     fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

StiffnessMatrixGPU::StiffnessMatrixGPU(Material& mat, Geometry &geo, unsigned int n)
  : StiffnessMatrixFirstOrder(mat, geo, n)
{
  int device = -1;
  hipGetDevice(&device);
  // copy from the material matarix
  hipMallocManaged(&D_d, 6*sizeof(double));
  hipMemcpy(D_d, material->materialMatrix, 6*sizeof(double), hipMemcpyHostToDevice);
  hipDeviceSynchronize();
  Log::Logger().Info("StiffnessMatrixGPU created by CPU");
};

StiffnessMatrixGPU::~StiffnessMatrixGPU()
{
  Log::Logger().Info("StiffnessMatrixGPU deleted by CPU");
  hipFree(D_d);
}

__global__ void constantCreatorKernel(int n, double* c, double* x, double* y, unsigned int* mesh, StiffnessMatrixGPU *s)
{
  //printf("in the function\n blockDim.x = %d, gridDim.x = %d, blockIdx.x = %d\n", blockDim.x,gridDim.x, blockIdx.x);
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    {
      //printf("i is %d stride is %d threadID = %d\n",i,stride,threadIdx.x);
      s->constantCreator(i, c, x, y, mesh);
    }
};


__global__ void StiffnessMatrixKernel(unsigned int n, unsigned int nip, double* in, unsigned int* ip, double* iw, double* c, double* D, unsigned int* mesh, double* k, unsigned int* i_index, unsigned int *j_index, unsigned int* dofFree,StiffnessMatrixGPU *obj)
{
  int index  = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    {
      obj->stiffnessMatrixCalculation(i, nip, in, ip, iw, c, D, mesh, k, i_index, j_index, dofFree);
    }
}

Sparse& StiffnessMatrixGPU::GetStiffnessMatrix()
{
  blockSize = 32;
  //numberOfElements=33;
  int numBlocks = (numberOfElements + blockSize-1)/blockSize;
  constantCreatorKernel<<<numBlocks, blockSize>>>(numberOfElements, c, geometry->get_x(), geometry->get_y(), geometry->get_mesh(), this);
  hipDeviceSynchronize();
  numBlocks = (simulationSize + blockSize-1)/blockSize;
  Timer timer("Time spend in GPU: ");
  StiffnessMatrixKernel<<<numBlocks, blockSize>>>(numberOfElements, nipSquared, integrationNode, integrationPos, integrationWeight, c, D_d, geometry->get_mesh(), stiffMat->value, stiffMat->i, stiffMat->j , geometry->get_Dof().get_free(),this);
  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  hipDeviceSynchronize();
  return *stiffMat;
}
